#include "hip/hip_runtime.h"
/* 
 * CUDA blur
 * Kevin Yuh, 2014 
 * Revised by Nailen Matschke, 2016
 * Revised by Loko Kung, 2018
 */

#include "blur.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>

#include "cuda_header.cuh"

CUDA_CALLABLE
void cuda_blur_kernel_convolution(uint thread_index, const float* gpu_raw_data,
                                  const float* gpu_blur_v, float* gpu_out_data,
                                  const unsigned int n_frames,
                                  const unsigned int blur_v_size) {
    // TODO: Implement the necessary convolution function that should be
    //       completed for each thread_index. Use the CPU implementation in
    //       blur.cpp as a reference.
    
    
    if(thread_index < blur_v_size)
    {
        for(int j=0; j <= thread_index; j++)
        {
            gpu_out_data[thread_index] += gpu_blur_v[j] * gpu_raw_data[thread_index - j];
        }
    }
    else
    {
        for(int j=0; j < blur_v_size; j++)
        {
            gpu_out_data[thread_index] += gpu_blur_v[j] * gpu_raw_data[thread_index - j];
        }
    }


}

__global__
void cuda_blur_kernel(const float *gpu_raw_data, const float *gpu_blur_v,
                      float *gpu_out_data, int n_frames, int blur_v_size) {
    // TODO: Compute the current thread index.
    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // TODO: Update the while loop to handle all indices for this thread.
    //       Remember to advance the index as necessary.
    while (thread_index < n_frames) {
        // Do computation for this thread index
        cuda_blur_kernel_convolution(thread_index, gpu_raw_data,
                                     gpu_blur_v, gpu_out_data,
                                     n_frames, blur_v_size);
        // TODO: Update the thread index
        thread_index += blockDim.x * gridDim.x;
    }
}


float cuda_call_blur_kernel(const unsigned int blocks,
                            const unsigned int threads_per_block,
                            const float *raw_data,
                            const float *blur_v,
                            float *out_data,
                            const unsigned int n_frames,
                            const unsigned int blur_v_size) {
    // Use the CUDA machinery for recording time
    hipEvent_t start_gpu, stop_gpu;
    float time_milli = -1;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);
    hipEventRecord(start_gpu);

    // TODO: Allocate GPU memory for the raw input data (either audio file
    //       data or randomly generated data. The data is of type float and
    //       has n_frames elements. Then copy the data in raw_data into the
    //       GPU memory you allocated.
    float* gpu_raw_data;

    // hipMalloc modifies this pointer to point to block of memory on device
    hipMalloc((void **) &gpu_raw_data, sizeof(float) * n_frames);
    hipMemcpy(gpu_raw_data, raw_data, sizeof(float) * n_frames, hipMemcpyHostToDevice);


    // TODO: Allocate GPU memory for the impulse signal (for now global GPU
    //       memory is fine. The data is of type float and has blur_v_size
    //       elements. Then copy the data in blur_v into the GPU memory you
    //       allocated.
    float* gpu_blur_v;
    hipMalloc((void **) &gpu_blur_v, sizeof(float) * blur_v_size);
    hipMemcpy(gpu_blur_v, blur_v, sizeof(float) * blur_v_size, hipMemcpyHostToDevice);


    
    // TODO: Allocate GPU memory to store the output audio signal after the
    //       convolution. The data is of type float and has n_frames elements.
    //       Initialize the data as necessary.
    float* gpu_out_data;
    hipMalloc((void **) &gpu_out_data, sizeof(float) * n_frames);
    hipMemset(gpu_out_data, 0, sizeof(float) * n_frames);
    
    // TODO: Appropriately call the kernel function.
    cuda_blur_kernel <<<blocks, threads_per_block>>> (
        gpu_raw_data,
        gpu_blur_v,
        gpu_out_data,
        n_frames,
        blur_v_size
    );


    // Check for errors on kernel call
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    else
        fprintf(stderr, "No kernel error detected\n");

    // TODO: Now that kernel calls have finished, copy the output signal
    //       back from the GPU to host memory. (We store this channel's result
    //       in out_data on the host.)
    hipMemcpy(out_data, gpu_out_data, sizeof(float) * n_frames, hipMemcpyDeviceToHost);
    hipFree(gpu_out_data);
    hipFree(gpu_blur_v);
    hipFree(gpu_raw_data);
    // TODO: Now that we have finished our computations on the GPU, free the
    //       GPU resources.

    // Stop the recording timer and return the computation time
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    hipEventElapsedTime(&time_milli, start_gpu, stop_gpu);
    return time_milli;
}
